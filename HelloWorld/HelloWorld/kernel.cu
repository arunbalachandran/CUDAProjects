#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
// the CUDA runtime is needed for __global__
using namespace std;

// kernel that the host will execute on the GPU
__global__ void AddIntegers(int *a, int *b)
{
	a[0] += b[0];
}

int main()
{
	int a = 7, b = 6;
	int *da, *db;
	// allocate memory for the device pointers
	hipMalloc(&da, sizeof(int));
	hipMalloc(&db, sizeof(int));
	// copy data from host to device
	hipMemcpy(da, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, &b, sizeof(int), hipMemcpyHostToDevice);
	// actual function call
	AddIntegers << <1, 1 >> >(da, db);
	// copy answer back to Host
	hipMemcpy(&a, da, sizeof(int), hipMemcpyDeviceToHost);
	cout << "The answer is " << a << endl;
	// free allocated memory on device
	hipFree(da);
	hipFree(db);
	return 0;
}